#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2023, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use 
 * under the terms of the LICENSE.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 */

#include "forward.h"
#include "auxiliary.h"
#include "ssr.h"
#include <hip/hip_cooperative_groups.h>
#include <math.h>
#include <cooperative_groups/reduce.h>
namespace cg = cooperative_groups;

// Forward method for converting the input spherical harmonics
// coefficients of each Gaussian to a simple RGB color.
__device__ glm::vec3 computeColorFromSH(
	const int idx,
	const int deg,
	const int max_coeffs,
	const glm::vec3* means,
	glm::vec3 campos,
	const float* shs,
	bool* clamped
) {
	// The implementation is loosely based on code for 
	// "Differentiable Point-Based Radiance Fields for 
	// Efficient View Synthesis" by Zhang et al. (2022)
	glm::vec3 pos = means[idx];
	glm::vec3 dir = pos - campos;
	dir = dir / glm::length(dir);

	glm::vec3* sh = ((glm::vec3*)shs) + idx * max_coeffs;
	glm::vec3 result = SH_C0 * sh[0];

	if (deg > 0)
	{
		float x = dir.x;
		float y = dir.y;
		float z = dir.z;
		result = result - SH_C1 * y * sh[1] + SH_C1 * z * sh[2] - SH_C1 * x * sh[3];

		if (deg > 1)
		{
			float xx = x * x, yy = y * y, zz = z * z;
			float xy = x * y, yz = y * z, xz = x * z;
			result = result +
				SH_C2[0] * xy * sh[4] +
				SH_C2[1] * yz * sh[5] +
				SH_C2[2] * (2.0f * zz - xx - yy) * sh[6] +
				SH_C2[3] * xz * sh[7] +
				SH_C2[4] * (xx - yy) * sh[8];

			if (deg > 2)
			{
				result = result +
					SH_C3[0] * y * (3.0f * xx - yy) * sh[9] +
					SH_C3[1] * xy * z * sh[10] +
					SH_C3[2] * y * (4.0f * zz - xx - yy) * sh[11] +
					SH_C3[3] * z * (2.0f * zz - 3.0f * xx - 3.0f * yy) * sh[12] +
					SH_C3[4] * x * (4.0f * zz - xx - yy) * sh[13] +
					SH_C3[5] * z * (xx - yy) * sh[14] +
					SH_C3[6] * x * (xx - 3.0f * yy) * sh[15];
			}
		}
	}
	result += 0.5f;

	// RGB colors are clamped to positive values. If values are
	// clamped, we need to keep track of this for the backward pass.
	clamped[3 * idx + 0] = (result.x < 0);
	clamped[3 * idx + 1] = (result.y < 0);
	clamped[3 * idx + 2] = (result.z < 0);
	return glm::max(result, 0.0f);
}

// Forward version of 2D covariance matrix computation
__device__ float3 computeCov2D(const float3& mean, float focal_x, float focal_y, float tan_fovx, float tan_fovy, const float* cov3D, const float* viewmatrix)
{
	// The following models the steps outlined by equations 29
	// and 31 in "EWA Splatting" (Zwicker et al., 2002). 
	// Additionally considers aspect / scaling of viewport.
	// Transposes used to account for row-/column-major conventions.
	float3 t = transformPoint4x3(mean, viewmatrix);

	const float limx = 1.3f * tan_fovx;
	const float limy = 1.3f * tan_fovy;
	const float txtz = t.x / t.z;
	const float tytz = t.y / t.z;
	t.x = min(limx, max(-limx, txtz)) * t.z;
	t.y = min(limy, max(-limy, tytz)) * t.z;

	glm::mat3 J = glm::mat3(
		focal_x / t.z, 0.0f, -(focal_x * t.x) / (t.z * t.z),
		0.0f, focal_y / t.z, -(focal_y * t.y) / (t.z * t.z),
		0, 0, 0);

	glm::mat3 W = glm::mat3(
		viewmatrix[0], viewmatrix[4], viewmatrix[8],
		viewmatrix[1], viewmatrix[5], viewmatrix[9],
		viewmatrix[2], viewmatrix[6], viewmatrix[10]);

	glm::mat3 T = W * J;

	glm::mat3 Vrk = glm::mat3(
		cov3D[0], cov3D[1], cov3D[2],
		cov3D[1], cov3D[3], cov3D[4],
		cov3D[2], cov3D[4], cov3D[5]);

	glm::mat3 cov = glm::transpose(T) * glm::transpose(Vrk) * T;

	// Apply low-pass filter: every Gaussian should be at least (equations 33)
	// one pixel wide/high. Discard 3rd row and column.
	cov[0][0] += 0.3f;
	cov[1][1] += 0.3f;
	return { float(cov[0][0]), float(cov[0][1]), float(cov[1][1]) };
}

// Forward method for converting scale and rotation properties of each
// Gaussian to a 3D covariance matrix in world space. Also takes care
// of quaternion normalization.
__device__ void computeCov3D(const glm::vec3 scale, float mod, const glm::vec4 rot, float* cov3D)
{
	// Create scaling matrix
	glm::mat3 S = glm::mat3(1.0f);
	S[0][0] = mod * scale.x;
	S[1][1] = mod * scale.y;
	S[2][2] = mod * scale.z;

	// Normalize quaternion to get valid rotation
	glm::vec4 q = rot;// / glm::length(rot);
	float r = q.x;
	float x = q.y;
	float y = q.z;
	float z = q.w;

	// Compute rotation matrix from quaternion
	glm::mat3 R = glm::mat3(
		1.f - 2.f * (y * y + z * z), 2.f * (x * y - r * z), 2.f * (x * z + r * y),
		2.f * (x * y + r * z), 1.f - 2.f * (x * x + z * z), 2.f * (y * z - r * x),
		2.f * (x * z - r * y), 2.f * (y * z + r * x), 1.f - 2.f * (x * x + y * y)
	);

	glm::mat3 M = S * R;

	// Compute 3D world covariance matrix Sigma
	glm::mat3 Sigma = glm::transpose(M) * M;

	// Covariance is symmetric, only store upper right
	cov3D[0] = Sigma[0][0];
	cov3D[1] = Sigma[0][1];
	cov3D[2] = Sigma[0][2];
	cov3D[3] = Sigma[1][1];
	cov3D[4] = Sigma[1][2];
	cov3D[5] = Sigma[2][2];
}

// Perform initial steps for each Gaussian prior to rasterization.
template<int C>
__global__ void preprocessCUDA(
	const int P, int D, int M,
	const float* orig_points,
	const glm::vec3* scales,
	const float scale_modifier,
	const glm::vec4* rotations,
	const float* opacities,
	const float* shs,
	const float* cov3D_precomp,
	const float* colors_precomp,
	const float* viewmatrix,
	const float* projmatrix,
	const glm::vec3* cam_pos,
	const int W, int H,
	const float tan_fovx, float tan_fovy,
	const float focal_x, float focal_y,
	bool* clamped,
	int* radii,
	float2* points_xy_image,
	float* depths,
	float3* pos_view,
	float* cov3Ds,
	float* rgb,
	float4* conic_opacity,
	uint32_t* tiles_touched,
	const dim3 grid,
	const bool prefiltered,
	const bool cubemap)
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= P)
		return;

	// Initialize radius and touched tiles to 0. If this isn't changed,
	// this Gaussian will not be processed further.
	radii[idx] = 0;
	tiles_touched[idx] = 0;

	// Perform near culling, quit if outside.
	float3 p_view;
	if (!in_frustum(idx, orig_points, viewmatrix, projmatrix, prefiltered, p_view))
		return;

	// Transform point by projecting
	const float3 p_orig = { orig_points[3 * idx], orig_points[3 * idx + 1], orig_points[3 * idx + 2] };
	const float4 p_hom = transformPoint4x4(p_orig, projmatrix);
	const float p_w = 1.0f / (p_hom.w + 0.0000001f);
	const float3 p_proj = { p_hom.x * p_w, p_hom.y * p_w, p_hom.z * p_w };

	// If 3D covariance matrix is precomputed, use it, otherwise compute
	// from scaling and rotation parameters. 
	const float* cov3D;
	if (cov3D_precomp != nullptr)
	{
		cov3D = cov3D_precomp + idx * 6;
	}
	else
	{
		computeCov3D(scales[idx], scale_modifier, rotations[idx], cov3Ds + idx * 6);
		cov3D = cov3Ds + idx * 6;
	}

	// Compute 2D screen-space covariance matrix
	const float3 cov = computeCov2D(p_orig, focal_x, focal_y, tan_fovx, tan_fovy, cov3D, viewmatrix);

	// Invert covariance (EWA algorithm)
	const float det = (cov.x * cov.z - cov.y * cov.y);
	if (det == 0.0f)
		return;
	const float det_inv = 1.f / det;
	const float3 conic = { cov.z * det_inv, -cov.y * det_inv, cov.x * det_inv };  // Inverse of cov2D

	// Compute extent in screen space (by finding eigenvalues of
	// 2D covariance matrix). Use extent to compute a bounding rectangle
	// of screen-space tiles that this Gaussian overlaps with. Quit if
	// rectangle covers 0 tiles. 
	const float mid = 0.5f * (cov.x + cov.z);
	const float lambda1 = mid + sqrt(max(0.1f, mid * mid - det));
	const float lambda2 = mid - sqrt(max(0.1f, mid * mid - det));
	const float my_radius = ceil(3.f * sqrt(max(lambda1, lambda2)));
	const float2 point_image = { ndc2Pix(p_proj.x, W), ndc2Pix(p_proj.y, H) };
	uint2 rect_min, rect_max;
	// Get the covered tile range by the point tile ids stored in `rect_min` and `rect_max`
	getRect(point_image, my_radius, grid, rect_min, rect_max);
	if ((rect_max.x - rect_min.x) * (rect_max.y - rect_min.y) == 0)
		return;

	// If colors have been precomputed, use them, otherwise convert
	// spherical harmonics coefficients to RGB color.
	if (colors_precomp == nullptr)
	{
		glm::vec3 result = computeColorFromSH(idx, D, M, (glm::vec3*)orig_points, *cam_pos, shs, clamped);
		rgb[idx * C + 0] = result.x;
		rgb[idx * C + 1] = result.y;
		rgb[idx * C + 2] = result.z;
	}

	// Store some useful helper data for the next steps.
	// if (cubemap) { // NOTE: To fix the discontinuity at the cubemap edges
	// 	const float3 dir = {p_orig.x - (*cam_pos).x, p_orig.y - (*cam_pos).y, p_orig.z - (*cam_pos).z};
	// 	depths[idx] = sqrtf(square_norm(dir));
	// } else {
	// 	depths[idx] = p_view.z;
	// }
	depths[idx] = p_view.z;
	radii[idx] = my_radius;
	points_xy_image[idx] = point_image;
	// Inverse 2D covariance and opacity neatly pack into one float4
	conic_opacity[idx] = { conic.x, conic.y, conic.z, opacities[idx] };
	pos_view[idx] = {p_view.x, p_view.y, p_view.z};
	tiles_touched[idx] = (rect_max.y - rect_min.y) * (rect_max.x - rect_min.x);	// The number of covered tiles
}


template <uint32_t CHANNELS>
__global__ void __launch_bounds__(BLOCK_X * BLOCK_Y)
liteRenderCUDA(
	const int W, int H,
	const uint2* __restrict__ ranges,
	const uint32_t* __restrict__ point_list,
	const float* __restrict__ features,
	const float2* __restrict__ points_xy_image,
	const float4* __restrict__ conic_opacity,
	const float* __restrict__ depth,
	const float* __restrict__ bg_color,
	uint32_t* __restrict__ n_contrib,
	float* __restrict__ final_T,
	float* __restrict__ out_color,
	float* __restrict__ out_opacity,
	float* __restrict__ out_depth,
	bool argmax_depth)
{
	// Identify current tile and associated min/max pixel range.
	auto block = cg::this_thread_block();
	uint32_t horizontal_blocks = (W + BLOCK_X - 1) / BLOCK_X;
	uint2 pix_min = { block.group_index().x * BLOCK_X, block.group_index().y * BLOCK_Y };
	uint2 pix_max = { min(pix_min.x + BLOCK_X, W), min(pix_min.y + BLOCK_Y , H) };
	uint2 pix = { pix_min.x + block.thread_index().x, pix_min.y + block.thread_index().y };
	uint32_t pix_id = W * pix.y + pix.x;
	float2 pixf = { (float)pix.x, (float)pix.y };

	// Check if this thread is associated with a valid pixel or outside.
	bool inside = pix.x < W && pix.y < H;
	// Done threads can help with fetching, but don't rasterize
	bool done = !inside;

	// Load start/end range of IDs to process in bit sorted list.
	uint2 range = ranges[block.group_index().y * horizontal_blocks + block.group_index().x];
	const int rounds = ((range.y - range.x + BLOCK_SIZE - 1) / BLOCK_SIZE);
	int toDo = range.y - range.x;

	// Allocate storage for batches of collectively fetched data.
	__shared__ int collected_id[BLOCK_SIZE];
	__shared__ float2 collected_xy[BLOCK_SIZE];
	__shared__ float4 collected_conic_opacity[BLOCK_SIZE];

	// Initialize helper variables
	float T = 1.0f;
	uint32_t contributor = 0;
	uint32_t last_contributor = 0;
	float C[CHANNELS] = { 0.0f };
	float D = 0.0f;
	float O = 0.0f;
	float max_weight = 0.0f;
	float except_depth = 0.0f;

	// Iterate over batches until all done or range is complete
	for (int i = 0; i < rounds; i++, toDo -= BLOCK_SIZE)
	{
		// End if entire block votes that it is done rasterizing
		int num_done = __syncthreads_count(done);
		if (num_done == BLOCK_SIZE)
			break;

		// Collectively fetch per-Gaussian data from global to shared
		int progress = i * BLOCK_SIZE + block.thread_rank();
		if (range.x + progress < range.y)
		{
			int coll_id = point_list[range.x + progress];
			collected_id[block.thread_rank()] = coll_id;
			collected_xy[block.thread_rank()] = points_xy_image[coll_id];
			collected_conic_opacity[block.thread_rank()] = conic_opacity[coll_id];
		}
		block.sync();

		// Iterate over current batch
		for (int j = 0; !done && j < min(BLOCK_SIZE, toDo); j++)
		{
			// Keep track of current position in range
			contributor++;

			// Resample using conic matrix (cf. "Surface 
			// Splatting" by Zwicker et al., 2001)
			float2 xy = collected_xy[j];
			float2 d = { xy.x - pixf.x, xy.y - pixf.y };
			float4 con_o = collected_conic_opacity[j];
			float power = -0.5f * (con_o.x * d.x * d.x + con_o.z * d.y * d.y) - con_o.y * d.x * d.y;
			if (power > 0.0f)
				continue;

			// Eq. (2) from 3D Gaussian splatting paper.
			// Obtain alpha by multiplying with Gaussian opacity
			// and its exponential falloff from mean.
			// Avoid numerical instabilities (see paper appendix). 
			float alpha = min(0.99f, con_o.w * exp(power));
			if (alpha < 1.0f / 255.0f)
				continue;
			float test_T = T * (1 - alpha);
			if (test_T < 0.0001f)
			{
				done = true;
				continue;
			}

			const float weight = alpha * T;
			// Eq. (3) from 3D Gaussian splatting paper.
			for (int ch = 0; ch < CHANNELS; ch++) {
				C[ch] += features[collected_id[j] * CHANNELS + ch] * weight;
			}

			D += depth[collected_id[j]] * weight;
			O += weight;

			// peak selection
			if (weight > max_weight) {
				except_depth = depth[collected_id[j]];
				max_weight = weight;
			}

			T = test_T;

			// Keep track of last range entry to update this
			// pixel.
			last_contributor = contributor;
		}
	}

	// All threads that treat valid pixel write out their final
	// rendering data to the frame and auxiliary buffers.
	if (inside)
	{
		final_T[pix_id] = T;
		n_contrib[pix_id] = last_contributor;
		for (int ch = 0; ch < CHANNELS; ch++) {
			out_color[ch * H * W + pix_id] = C[ch] + T * bg_color[ch];
		}
		if (O > 1e-6) {
			out_depth[pix_id] = argmax_depth ? except_depth : D / O; // peak selection or linear interpolation
		} else {
			out_depth[pix_id] = 0.0f;
		}
		out_opacity[pix_id] = O;
	}
}

// Main rasterization method. Collaboratively works on one tile per
// block, each thread treats one pixel. Alternates between fetching 
// and rasterizing data.
template <uint32_t CHANNELS>
__global__ void __launch_bounds__(BLOCK_X * BLOCK_Y)
renderCUDA(
	const int W, int H,
	const float fx, float fy,
	const float* means3D,
	const float* cam_pos,
	const uint2* __restrict__ ranges,
	const uint32_t* __restrict__ point_list,
	const float* viewmatrix,
	const float* __restrict__ features,
	const float* __restrict__ normals,
	const float* __restrict__ albedo,
	const float* __restrict__ roughness,
	const float* __restrict__ metallic,
	const float* __restrict__ semantic,		// [P, 20] // 新增
	const float* __restrict__ flow,			// [P, 2] // 新增
	const float3* __restrict__ pos_view,
	const float2* __restrict__ points_xy_image,
	const float4* __restrict__ conic_opacity,
	const float* __restrict__ depth,
	const float* __restrict__ bg_color,
	uint32_t* __restrict__ n_contrib,
	float* __restrict__ final_T,
	float* __restrict__ out_color,
	float* __restrict__ out_opacity,
	float* __restrict__ out_depth,
	float* __restrict__ out_normal,
	float* __restrict__ out_normal_view,
	float* __restrict__ out_pos,
	float* __restrict__ out_albedo,
	float* __restrict__ out_roughness,
	float* __restrict__ out_metallic,
	float* __restrict__ out_semantic,		// [20, H, W] // 新增
	float* __restrict__ out_flow,			// [2, H, W] //	
	bool argmax_depth,
	bool inference)
{
	// Identify current tile and associated min/max pixel range.
	auto block = cg::this_thread_block();
	uint32_t horizontal_blocks = (W + BLOCK_X - 1) / BLOCK_X;
	uint2 pix_min = { block.group_index().x * BLOCK_X, block.group_index().y * BLOCK_Y };
	uint2 pix_max = { min(pix_min.x + BLOCK_X, W), min(pix_min.y + BLOCK_Y , H) };
	uint2 pix = { pix_min.x + block.thread_index().x, pix_min.y + block.thread_index().y };
	uint32_t pix_id = W * pix.y + pix.x;
	float2 pixf = { (float)pix.x, (float)pix.y };
	float cx = float(W) / 2.0f, cy = float(H) / 2.0f;
	const float3 ray = {(pixf.x - cx) / fx, (pixf.y - cy) / fy, 1.0f};

	// Check if this thread is associated with a valid pixel or outside.
	bool inside = pix.x < W && pix.y < H;
	// Done threads can help with fetching, but don't rasterize
	bool done = !inside;

	// Load start/end range of IDs to process in bit sorted list.
	uint2 range = ranges[block.group_index().y * horizontal_blocks + block.group_index().x];
	const int rounds = ((range.y - range.x + BLOCK_SIZE - 1) / BLOCK_SIZE);
	int toDo = range.y - range.x;

	// Allocate storage for batches of collectively fetched data.
	__shared__ int collected_id[BLOCK_SIZE];
	__shared__ float2 collected_xy[BLOCK_SIZE];
	__shared__ float4 collected_conic_opacity[BLOCK_SIZE];

	// Initialize helper variables
	float T = 1.0f;
	uint32_t contributor = 0;
	uint32_t last_contributor = 0;
	float C[CHANNELS] = { 0.0f };
	float N[CHANNELS] = { 0.0f };
	float A[CHANNELS] = { 0.0f };
	float R = 0.0f;
	float M = 0.0f;
	float D = 0.0f;
	float S[20] =  { 0.0f }; // 新增
	float F[2] = { 0.0f }; // 

	float3 POS = {0.0f, 0.0f, 0.0f};
	float3 N_world = {0.0f, 0.0f, 0.0f};
	float3 N_view = {0.0f, 0.0f, 0.0f};
	float O = 0.0f;
	float max_weight = 0.0f;
	float except_depth = 0.0f;
	float3 except_pos = {0.0f, 0.0f, 0.0f};


	// Iterate over batches until all done or range is complete
	for (int i = 0; i < rounds; i++, toDo -= BLOCK_SIZE)
	{
		// End if entire block votes that it is done rasterizing
		int num_done = __syncthreads_count(done);
		if (num_done == BLOCK_SIZE)
			break;

		// Collectively fetch per-Gaussian data from global to shared
		int progress = i * BLOCK_SIZE + block.thread_rank();
		if (range.x + progress < range.y)
		{
			int coll_id = point_list[range.x + progress];  // Get the idx of 3D Gaussian
			collected_id[block.thread_rank()] = coll_id;
			collected_xy[block.thread_rank()] = points_xy_image[coll_id];
			collected_conic_opacity[block.thread_rank()] = conic_opacity[coll_id];
		}
		block.sync();

		// Iterate over current batch
		for (int j = 0; !done && j < min(BLOCK_SIZE, toDo); j++)
		{
			// Keep track of current position in range
			contributor++;

			// Resample using conic matrix (cf. "Surface 
			// Splatting" by Zwicker et al., 2001)
			float2 xy = collected_xy[j];
			float2 d = { xy.x - pixf.x, xy.y - pixf.y };
			float4 con_o = collected_conic_opacity[j];
			float power = -0.5f * (con_o.x * d.x * d.x + con_o.z * d.y * d.y) - con_o.y * d.x * d.y;
			if (power > 0.0f)
				continue;

			// Eq. (2) from 3D Gaussian splatting paper.
			// Obtain alpha by multiplying with Gaussian opacity
			// and its exponential falloff from mean.
			// Avoid numerical instabilities (see paper appendix). 
			float alpha = min(0.99f, con_o.w * exp(power));
			if (alpha < 1.0f / 255.0f)
				continue;
			float test_T = T * (1 - alpha);
			if (test_T < 0.0001f) {
				done = true;
				continue;
			}

			float3 view_dir = {
				cam_pos[0] - means3D[collected_id[j] * 3 + 0],
				cam_pos[1] - means3D[collected_id[j] * 3 + 1],
				cam_pos[2] - means3D[collected_id[j] * 3 + 2],
			};
			const float NoV = normals[collected_id[j] * 3 + 0] * view_dir.x + \
							  normals[collected_id[j] * 3 + 1] * view_dir.y + \
							  normals[collected_id[j] * 3 + 2] * view_dir.z;

			const float weight = alpha * T;
			// Eq. (3) from 3D Gaussian splatting paper.
			for (int ch = 0; ch < CHANNELS; ch++) {
				C[ch] += features[collected_id[j] * CHANNELS + ch] * weight;
				A[ch] += albedo[collected_id[j] * CHANNELS + ch] * weight;
                //if (NoV > 0.0f) // NOTE: the trick from GIR, do not make scene for scenes
				N[ch] += normals[collected_id[j] * CHANNELS + ch] * weight;
			}
			R += roughness[collected_id[j]] * weight;
			M += metallic[collected_id[j]] * weight;
			
			//语义渲染
			for (int ch = 0; ch < 20; ch++) {
				S[ch] += semantic[collected_id[j] * 20 + ch] * weight;
			}
			//flow
			for (int ch = 0; ch < 2; ch++) {
				F[ch] += flow[collected_id[j] * 2 + ch] * weight;
			}

			D += depth[collected_id[j]] * weight;
			POS.x += pos_view[collected_id[j]].x * weight;
			POS.y += pos_view[collected_id[j]].y * weight;
			POS.z += pos_view[collected_id[j]].z * weight;
			O += weight;

			if (weight > max_weight) {
				except_depth = depth[collected_id[j]];
				except_pos.x = pos_view[collected_id[j]].x;
				except_pos.y = pos_view[collected_id[j]].y;
				except_pos.z = pos_view[collected_id[j]].z;
				max_weight = weight;
			}

			T = test_T;

			// Keep track of last range entry to update this
			// pixel.
			last_contributor = contributor;
		}
	}

	// All threads that treat valid pixel write out their final
	// rendering data to the frame and auxiliary buffers.
	if (inside)
	{
		final_T[pix_id] = T;
		n_contrib[pix_id] = last_contributor;

		N_world = {N[0], N[1], N[2]};
		N_view = transformVec4x3(N_world, viewmatrix);
		N_view = normalize(N_view);
		out_normal_view[pix_id] = N_view.x;
		out_normal_view[1 * H * W + pix_id] = N_view.y;
		out_normal_view[2 * H * W + pix_id] = N_view.z;
		
		for (int ch = 0; ch < CHANNELS; ch++) {
			out_color[ch * H * W + pix_id] = C[ch] + T * bg_color[ch];
			out_normal[ch * H * W + pix_id] = N[ch];
			out_albedo[ch * H * W + pix_id] = A[ch];
		}
		if (inference) {
			out_roughness[pix_id] = R + T;
		} else {
			out_roughness[pix_id] = R;
		}
		out_metallic[pix_id] = M;

		//语义
		for (int ch = 0; ch < 20; ch++) {
			out_semantic[ch * H * W + pix_id] = S[ch];
		}
		//flow
		out_flow[0 * H * W + pix_id] = F[0];
		out_flow[1 * H * W + pix_id] = F[1];

		if (O > 1e-6) {
			out_depth[pix_id] = argmax_depth ? except_depth : D / O;
			out_pos[pix_id] = argmax_depth ? except_pos.x : POS.x / O;
			out_pos[1 * H * W + pix_id] = argmax_depth ? except_pos.y : POS.y / O;
			out_pos[2 * H * W + pix_id] = argmax_depth ? except_pos.z : POS.z / O;
		} else {
			out_depth[pix_id] = 0.0f;
			out_pos[pix_id] = 0.0f;
			out_pos[1 * H * W + pix_id] = 0.0f;
			out_pos[2 * H * W + pix_id] = 0.0f;
		}
		

		out_opacity[pix_id] = O;
	}
}

__global__ void __launch_bounds__(BLOCK_X * BLOCK_Y)
SSAOCUDA(
	int W, int H,
	const float focal_x,
	const float focal_y,
	const float radius,  //0.8
	const float bias, //-0.01
	const float thick, //-0.05
	const float delta, //0.0625
	const int step, //16
	const int start, //8
	const float* __restrict__ out_normal,
	const float* __restrict__ out_pos,
	float* __restrict__ occlusion)
{
	auto block = cg::this_thread_block();
	uint2 pix_min = { block.group_index().x * BLOCK_X, block.group_index().y * BLOCK_Y };
	uint2 pix_max = { min(pix_min.x + BLOCK_X, W), min(pix_min.y + BLOCK_Y , H) };
	uint2 pix = { pix_min.x + block.thread_index().x, pix_min.y + block.thread_index().y };
	uint32_t pix_id = W * pix.y + pix.x;
	if (pix.x > W-1 || pix.y > H-1)
		return;

	float3 normal_un = {out_normal[pix_id], out_normal[1 * H * W + pix_id], out_normal[2 * H * W + pix_id]};
	float3 normal = normalize(normal_un);
	float3 pos = {out_pos[pix_id], out_pos[1 * H * W + pix_id], out_pos[2 * H * W + pix_id]};
	float3 up = {0.0f, 1.0f, 0.0f};
	float rndot = dot(up, normal); 
	float3 untangent = {up.x - normal.x * rndot, up.y - normal.y * rndot, up.z - normal.z * rndot};
	float3 tangent = normalize(untangent);
	float3 bitangent = normalize(cross(normal, tangent));
	float TBN[9];
	TBN[0] = tangent.x;
	TBN[1] = tangent.y;
	TBN[2] = tangent.z;
	TBN[3] = bitangent.x;
	TBN[4] = bitangent.y;
	TBN[5] = bitangent.z;
	TBN[6] = normal.x;
	TBN[7] = normal.y;
	TBN[8] = normal.z;
	float occ = 0.0;
	float sampleDelta = delta * M_PIf;
    float nrSamples = 0.0; 
    for(float phi = 0.0; phi < 2.0 * M_PIf; phi += sampleDelta)
    {
        for(float theta = 0.0; theta <= 0.5 * M_PIf; theta += sampleDelta * 0.5)
        {
        // spherical to cartesian (in tangent space)
			float cosh = cosf(theta);
            float3 tangentSample = {sinf(theta) * cosf(phi),  sinf(theta) * sinf(phi), cosf(theta)};
            tangentSample = normalize(tangentSample);
        // tangent space to view
            float3 sampleVec = transformVec3x3(tangentSample, TBN);
            float3 samplePos = {0.0f, 0.0f, 0.0f};
			nrSamples += cosh * sinf(theta);
		    for(int j = start; j < step; ++j)
		    {
			    samplePos.x = pos.x + sampleVec.x * j * (1 + pos.z / 100) * (1 + pos.z / 100 ) * radius / step; //100=zfar-znear
			    samplePos.y = pos.y + sampleVec.y * j * (1 + pos.z / 100) * (1 + pos.z / 100)* radius / step; 
			    samplePos.z = pos.z + sampleVec.z * j * (1 + pos.z / 100) * (1 + pos.z / 100) * radius / step; 
			    float cx = float(W) / 2.0f, cy = float(H) / 2.0f;
			    int2 depth_id = get_coord(cx, cy, focal_x, focal_y, samplePos);
			    if (depth_id.x < 0)
				    break;
			    else if (depth_id.x > W - 1)
				    break;
			    if (depth_id.y < 0)
				    break;
			    else if (depth_id.y > H - 1)
				    break;
				float sampleDepth = out_pos[2 * H * W + W * depth_id.y + depth_id.x]; 
				

			    if (sampleDepth <= samplePos.z + bias && sampleDepth >= samplePos.z - thick) 
			    {
				    occ += cosh * sinf(theta);
				    break;
			    }
		    }
        }
    }
	if(nrSamples > 0.0){
		occlusion[pix_id] = fmaxf(0.0f, fminf(1.0f, 1.0 - (occ / nrSamples)));

	}
    else{
		occlusion[pix_id] = 1.0;
	}
}

__global__ void __launch_bounds__(BLOCK_X * BLOCK_Y)
SSRCUDA(
	int W, int H,
	const float focal_x,
	const float focal_y,
	const float radius,  //0.8
	const float bias, //-0.01
	const float thick, //-0.05
	const float delta, //0.0625
	const int step, //16
	const int start, //8
	const float* __restrict__ out_normal,
	const float* __restrict__ out_pos,
	const float* __restrict__ out_rgb,
    const float* __restrict__ out_albedo,
    const float* __restrict__ out_roughness,
    const float* __restrict__ out_metallic,
    const float* __restrict__ out_F0,
	float* __restrict__ color,
	float* __restrict__ abd)
{
	auto block = cg::this_thread_block();
	uint2 pix_min = { block.group_index().x * BLOCK_X, block.group_index().y * BLOCK_Y };
	uint2 pix_max = { min(pix_min.x + BLOCK_X, W), min(pix_min.y + BLOCK_Y , H) };
	uint2 pix = { pix_min.x + block.thread_index().x, pix_min.y + block.thread_index().y };
	uint32_t pix_id = W * pix.y + pix.x;
	if (pix.x > W-1 || pix.y > H-1)
		return;
    float3 pos = {out_pos[pix_id], out_pos[1 * H * W + pix_id], out_pos[2 * H * W + pix_id]};

    float3 diffuse = {0.0f, 0.0f, 0.0f}; 
	float3 gd = {0.0f, 0.0f, 0.0f}; 
    float3 specular = {0.0f, 0.0f, 0.0f};                                                                                                 
	float3 normal_un = {out_normal[pix_id], out_normal[1 * H * W + pix_id], out_normal[2 * H * W + pix_id]};
	float3 normal = normalize(normal_un);
    float3 N = normal;
    float3 up = {0.0f, 1.0f, 0.0f};
    float rndot = dot(up, normal); 
	float3 untangent = {up.x - normal.x * rndot, up.y - normal.y * rndot, up.z - normal.z * rndot};
	float3 tangent = normalize(untangent);
	float3 bitangent = normalize(cross(normal, tangent));
    float TBN[9];
	TBN[0] = tangent.x;
	TBN[1] = tangent.y;
	TBN[2] = tangent.z;
	TBN[3] = bitangent.x;
	TBN[4] = bitangent.y;
	TBN[5] = bitangent.z;
	TBN[6] = normal.x;
	TBN[7] = normal.y;
	TBN[8] = normal.z;


    float3 albedo = {out_albedo[pix_id], out_albedo[1 * H * W + pix_id], out_albedo[2 * H * W + pix_id]};
    float3 F0 = {out_F0[pix_id], out_F0[1 * H * W + pix_id], out_F0[2 * H * W + pix_id]};
    float roughness = out_roughness[pix_id];
    float metallic = out_metallic[pix_id];

    float3 V = normalize(-pos);
    float3 F = fresnelSchlick(fmaxf(dot(N, V), 0.0000001), F0);
    float3 kS = F;
    float3 kD = {1.0 - kS.x, 1.0 - kS.y, 1.0 - kS.z};
    kD.x *= 1.0 - metallic;
    kD.y *= 1.0 - metallic;
    kD.z *= 1.0 - metallic;

    float sampleDelta = delta * M_PIf;
    float nrSamples = 0.0; 
    for(float phi = 0.0; phi < 2.0 * M_PIf; phi += sampleDelta)
    {
        for(float theta = 0.0; theta <= 0.5 * M_PIf; theta += sampleDelta * 0.5)
        {
        // spherical to cartesian (in tangent space)
            float3 tangentSample = {sinf(theta) * cosf(phi),  sinf(theta) * sinf(phi), cosf(theta)};
            tangentSample = normalize(tangentSample);
        // tangent space to view
            float3 sampleVec = transformVec3x3(tangentSample, TBN);
            float3 samplePos = {0.0f, 0.0f, 0.0f};
			nrSamples += 1;
		    for(int j = start; j < step; ++j)
		    {
			    samplePos.x = pos.x + sampleVec.x * j * (1 + pos.z / 100) * (1 + pos.z / 100) * radius / step; 
			    samplePos.y = pos.y + sampleVec.y * j * (1 + pos.z / 100) * (1 + pos.z / 100)* radius / step; 
			    samplePos.z = pos.z + sampleVec.z * j * (1 + pos.z / 100) * (1 + pos.z / 100) * radius / step; 
			    float cx = float(W) / 2.0f, cy = float(H) / 2.0f;
			    int2 depth_id = get_coord(cx, cy, focal_x, focal_y, samplePos);
			    if (depth_id.x < 0)
				    break;
			    else if (depth_id.x > W - 1)
				    break;
			    if (depth_id.y < 0)
				    break;
			    else if (depth_id.y > H - 1)
				    break;
			    float3 rgb = {out_rgb[W * depth_id.y + depth_id.x], out_rgb[H * W + W * depth_id.y + depth_id.x], out_rgb[2 * H * W + W * depth_id.y + depth_id.x]}; 
				float sampleDepth = out_pos[2 * H * W + W * depth_id.y + depth_id.x]; 
			    if (sampleDepth <= samplePos.z + bias && sampleDepth >= samplePos.z - thick)  //0.05 0.1
			    {
				    diffuse.x += rgb.x * cosf(theta) * sinf(theta);
                    diffuse.y += rgb.y * cosf(theta) * sinf(theta);
                    diffuse.z += rgb.z * cosf(theta) * sinf(theta);
                    // nrSamples++;
				    break;
			    }
		    }
        }
    }
	if(nrSamples > 0.0){
		gd.x = M_PIf * diffuse.x * (1.0 / float(nrSamples)) * kD.x;  //calculate gradient in forward pass, no backward.
		gd.y = M_PIf * diffuse.y * (1.0 / float(nrSamples)) * kD.y;
    	gd.z = M_PIf * diffuse.z * (1.0 / float(nrSamples)) * kD.z;
		diffuse.x = gd.x * albedo.x;
		diffuse.y = gd.y * albedo.y;
    	diffuse.z = gd.z * albedo.z;
	}
    else{
		diffuse.x = 0.0000001;
		diffuse.y = 0.0000001;
    	diffuse.z = 0.0000001;
		gd.x = 0.0000001;
		gd.y = 0.0000001;
    	gd.z = 0.0000001;
	}
   

//------------------------- indlight for specular component, you can modify it if you are interested------//
	// nrSamples = 0.0; 
    // const uint SAMPLE_COUNT = 64;      
    // for(uint i = 0u; i < SAMPLE_COUNT; ++i)
    // {
    //     float3 samplePos = {0.0f, 0.0f, 0.0f};
    //     float2 Xi = Hammersley(i, SAMPLE_COUNT);
    //     float3 Half = ImportanceSampleGGX(Xi, N, roughness);
    //     float3 L = normalize(2.0 * dot(V, Half) * Half - V);
    //     float NdotL = fmaxf(dot(N, L), 0.0);
    //     if(NdotL > 0.0)
    //     {
    //         for(int j = 4; j < step; ++j)
	// 	    {
	// 		    samplePos.x = pos.x + L.x * j * (1 + pos.z / 100) * (1 + pos.z / 100 ) * radius / step; 
	// 		    samplePos.y = pos.y + L.y * j * (1 + pos.z / 100) * (1 + pos.z / 100)* radius / step; 
	// 		    samplePos.z = pos.z + L.z * j * (1 + pos.z / 100) * (1 + pos.z / 100) * radius / step; 
	// 		    float cx = float(W) / 2.0f, cy = float(H) / 2.0f;
	// 		    int2 depth_id = get_coord(cx, cy, focal_x, focal_y, samplePos);
	// 		    if (depth_id.x < 0)
	// 			    break;
	// 		    else if (depth_id.x > W - 1)
	// 			    break;
	// 		    if (depth_id.y < 0)
	// 			    break;
	// 		    else if (depth_id.y > H - 1)
	// 			    break;
	// 		    float3 rgb = {out_rgb[W * depth_id.y + depth_id.x], out_rgb[H * W + W * depth_id.y + depth_id.x], out_rgb[2 * H * W + W * depth_id.y + depth_id.x]}; 
	// 			float sampleDepth = out_pos[2 * H * W + W * depth_id.y + depth_id.x]; 
	// 		    if (sampleDepth <= samplePos.z + bias && sampleDepth >= samplePos.z - 0.15)
	// 		    {
				    
    //                 float attenuation = 1.0 / ((samplePos.x-pos.x)*(samplePos.x-pos.x)+(samplePos.y-pos.y)*(samplePos.y-pos.y)+(samplePos.z-pos.z)*(samplePos.z-pos.z)+0.0001);
    //                 float3 radiance = {rgb.x * attenuation, rgb.y * attenuation, rgb.z * attenuation};
    //                 float NDF = DistributionGGX(N, Half, roughness);        
    //                 float G = GeometrySmith(N, V, L, roughness);      
    //                 float3 nominator = {NDF * G * F.x, NDF * G * F.y, NDF * G * F.z};
    //                 float denominator = 4.0 * fmaxf(dot(N, V), 0.0) * fmaxf(dot(N, L), 0.0) + 0.001; 
    //                 float3 spec = {nominator.x / denominator, nominator.y / denominator, nominator.z / denominator};               
    //                 specular.x += spec.x * radiance.x * NdotL; 
    //                 specular.y += spec.y * radiance.y * NdotL; 
    //                 specular.z += spec.z * radiance.z * NdotL; 
	// 				nrSamples++;
	// 			    break;
	// 		    }
	// 	    }
    //     }
    // }
    color[pix_id] = diffuse.x;
    color[1 * H * W + pix_id] = diffuse.y;
    color[2 * H * W + pix_id] = diffuse.z;	

	abd[pix_id] = gd.x;
    abd[1 * H * W + pix_id] = gd.y;
    abd[2 * H * W + pix_id] = gd.z;
	// color[pix_id] = diffuse.x + specular.x * (1.0 / float(nrSamples));
    // color[1 * H * W + pix_id] = diffuse.y + specular.y * (1.0 / float(nrSamples));
    // color[2 * H * W + pix_id] = diffuse.z + specular.z * (1.0 / float(nrSamples));	
}




__global__ void __launch_bounds__(BLOCK_X * BLOCK_Y)
depthmapToNormalCUDA(
	int W, int H,
	const float focal_x,
	const float focal_y,
	const float* __restrict__ viewmatrix,
	const float* __restrict__ out_depth,
	float* __restrict__ normal_from_depth,
	float* __restrict__ depth_pos)
{
	// Identify current tile and associated min/max pixel range.

	
	auto block = cg::this_thread_block();
	uint2 pix_min = { block.group_index().x * BLOCK_X, block.group_index().y * BLOCK_Y };
	uint2 pix_max = { min(pix_min.x + BLOCK_X, W), min(pix_min.y + BLOCK_Y , H) };
	uint2 pix = { pix_min.x + block.thread_index().x, pix_min.y + block.thread_index().y };
	uint32_t pix_id = W * pix.y + pix.x;

	// if (pix.x > W-1 || pix.y > H-1)
	// 	return;
	// Check if this thread is associated with a valid pixel or outside.
	if (pix.x <= 0 || pix.x >= W - 1 || pix.y <= 0 || pix.y >= H - 1) return;


	const float depth_thresh = 0.01f;
	const float depth = out_depth[pix_id];
	float cx = float(W) / 2.0f, cy = float(H) / 2.0f;
	float3 pos = get_position(pix.x, pix.y, cx, cy, focal_x, focal_y, depth);
	depth_pos[pix_id] = pos.x;
	depth_pos[1 * H * W + pix_id] = pos.y;
	depth_pos[2 * H * W + pix_id] = pos.z;

	if (depth < depth_thresh) return;

	// int pad = 2;
	// for (int x = -pad; x < pad + 1; ++x) {
	// 	if (int(pix.x + x) < 0 || int(pix.x + x) > W - 1) return;
	// 	for (int y = -pad; y < pad + 1; ++y) {
	// 		if (int(pix.y + y) < 0 || int(pix.y + y) > H - 1) return;
	// 		if (out_depth[pix_id + y + W * x] < depth_thresh) return;
	// 	}
	// }
	// float depth_left = out_depth[pix_id - 1], depth_right = out_depth[pix_id + 1];
	// float depth_up = out_depth[pix_id - W], depth_down = out_depth[pix_id + W];

	// float3 pos_cen = get_position(pix.x, pix.y, cx, cy, focal_x, focal_y, depth);
	// float3 pos_left = get_position(pix.x - 1, pix.y, cx, cy, focal_x, focal_y, depth_left);
	// float3 pos_right = get_position(pix.x + 1, pix.y, cx, cy, focal_x, focal_y, depth_right);
	// float3 pos_up = get_position(pix.x, pix.y - 1, cx, cy, focal_x, focal_y, depth_up);
	// float3 pos_down = get_position(pix.x, pix.y + 1, cx, cy, focal_x, focal_y, depth_down);
	// float3 ddx = fabsf(depth_left - depth) < fabs(depth_right - depth) ? (pos_cen - pos_left) : (pos_right - pos_cen);
	// float3 ddy = fabsf(depth_down - depth) < fabs(depth_up - depth) ? (pos_cen - pos_down) : (pos_up - pos_cen);
	// float3 normal = cross(ddx, ddy);
	// normal = normalize(normal);

	// // NOTE: rotation (it should be c2w!!!)
	// const float normal_x = viewmatrix[0] * normal.x + viewmatrix[1] * normal.y + viewmatrix[2] * normal.z;
	// const float normal_y = viewmatrix[4] * normal.x + viewmatrix[5] * normal.y + viewmatrix[6] * normal.z;
	// const float normal_z = viewmatrix[8] * normal.x + viewmatrix[9] * normal.y + viewmatrix[10] * normal.z;

	// normal_from_depth[0 * H * W + pix_id] = normal_x;
	// normal_from_depth[1 * H * W + pix_id] = normal_y;
	// normal_from_depth[2 * H * W + pix_id] = normal_z;
	// filter out the edge to avoid the noise normal
	int pad = 2;
	for (int x = -pad; x < pad + 1; ++x) {
		if (int(pix.x + x) < 0 || int(pix.x + x) > W - 1) return;
		for (int y = -pad; y < pad + 1; ++y) {
			if (int(pix.y + y) < 0 || int(pix.y + y) > H - 1) return;
			if (out_depth[pix_id + W * y + x] < depth_thresh) return;
		}
	}
	float depth_aa = out_depth[pix_id - W];
	float depth_bb = out_depth[pix_id + 1];
	float depth_cc = out_depth[pix_id + W];
	float depth_dd = out_depth[pix_id - 1];
	float depth_ab = out_depth[pix_id - W + 1];
	float depth_bc = out_depth[pix_id + W + 1];
	float depth_cd = out_depth[pix_id + W - 1];
	float depth_da = out_depth[pix_id - W - 1];

	float3 pos_aa = get_position(pix.x, pix.y - 1, cx, cy, focal_x, focal_y, depth_aa);
	float3 pos_bb = get_position(pix.x + 1, pix.y, cx, cy, focal_x, focal_y, depth_bb);
	float3 pos_cc = get_position(pix.x, pix.y + 1, cx, cy, focal_x, focal_y, depth_cc);
	float3 pos_dd = get_position(pix.x - 1, pix.y, cx, cy, focal_x, focal_y, depth_dd);
	float3 pos_ab = get_position(pix.x + 1, pix.y - 1, cx, cy, focal_x, focal_y, depth_ab);
	float3 pos_bc = get_position(pix.x + 1, pix.y + 1, cx, cy, focal_x, focal_y, depth_bc);
	float3 pos_cd = get_position(pix.x - 1, pix.y + 1, cx, cy, focal_x, focal_y, depth_cd);
	float3 pos_da = get_position(pix.x - 1, pix.y - 1, cx, cy, focal_x, focal_y, depth_da);
	float3 edge_a = pos_da - pos_ab;
	float3 edge_b = pos_ab - pos_bc;
	float3 edge_c = pos_bc - pos_cd;
	float3 edge_d = pos_cd - pos_da;
	float3 edge_ac = pos_cc - pos_aa;
	float3 edge_bd = pos_dd - pos_bb;
	float3 edge_cdab = pos_ab - pos_cd;
	float3 edge_bcad = pos_da - pos_bc;

	float3 normal1 = cross(edge_a, edge_d);
	float3 normal2 = cross(edge_d, edge_c);
	float3 normal3 = cross(edge_c, edge_b);
	float3 normal4 = cross(edge_b, edge_a);
	float3 normal5 = cross(edge_ac, edge_bd);
	float3 normal6 = cross(edge_bcad, edge_cdab);
	float3 normal = (normalize(normal1) + normalize(normal2) + normalize(normal3) + normalize(normal4) + normalize(normal5) + normalize(normal6))/6;

	// NOTE: rotation (it should be c2w!!!)
	const float normal_x = viewmatrix[0] * normal.x + viewmatrix[1] * normal.y + viewmatrix[2] * normal.z;
	const float normal_y = viewmatrix[4] * normal.x + viewmatrix[5] * normal.y + viewmatrix[6] * normal.z;
	const float normal_z = viewmatrix[8] * normal.x + viewmatrix[9] * normal.y + viewmatrix[10] * normal.z;
	// const float normal_x = normal.x;
	// const float normal_y = normal.y;
	// const float normal_z = normal.z;

	normal_from_depth[pix_id] = normal_x;
	normal_from_depth[1 * H * W + pix_id] = normal_y;
	normal_from_depth[2 * H * W + pix_id] = normal_z;
}

void FORWARD::lite_render(
	const dim3 grid, dim3 block,
	int W, int H,
	const uint2* ranges,
	const uint32_t* point_list,
	const float* colors,
	const float2* means2D,
	const float4* conic_opacity,
	const float* depth,
	const float* bg_color,
	uint32_t* n_contrib,
	float* final_T,
	float* out_color,
	float* out_opacity,
	float* out_depth,
	bool argmax_depth)
{
	liteRenderCUDA<NUM_CHANNELS><<<grid, block>>>(
		W, H,
		ranges,
		point_list,
		colors,
		means2D,
		conic_opacity,
		depth,
		bg_color,
		n_contrib,
		final_T,
		out_color,
		out_opacity,
		out_depth,
		argmax_depth);
}

void FORWARD::render(
	const dim3 grid, dim3 block,
	const int W, int H,
	const float fx, float fy,
	const float* means3D,
	const float* cam_pos,
	const uint2* ranges,
	const uint32_t* point_list,
	const float* viewmatrix,
	const float* colors,
	const float* normal,
	const float* albedo,
	const float* roughness,
	const float* metallic,
	const float* semantic,// semantic
	const float* flow,
	const float3* pos_view,
	const float2* means2D,
	const float4* conic_opacity,
	const float* depth,
	const float* bg_color,
	uint32_t* n_contrib,
	float* final_T,
	float* out_color,
	float* out_opacity,
	float* out_depth,
	float* out_normal,
	float* out_normal_view,
	float* out_pos,
	float* out_albedo,
	float* out_roughness,
	float* out_metallic,
	float* out_semantic,
	float* out_flow, // 新增
	const bool argmax_depth,
	const bool inference)
{
	renderCUDA<NUM_CHANNELS><<<grid, block>>>(
		W, H,
		fx, fy,
		means3D,
		cam_pos,
		ranges,
		point_list,
		viewmatrix,
		colors,
		normal,
		albedo,
		roughness,
		metallic,
		semantic,
		flow,
		pos_view,
		means2D,
		conic_opacity,
		depth,
		bg_color,
		n_contrib,
		final_T,
		out_color,
		out_opacity,
		out_depth,
		out_normal,
		out_normal_view, 
		out_pos,
		out_albedo,
		out_roughness,
		out_metallic,
		out_semantic,
		out_flow, // 新增
		argmax_depth,
		inference);
}

void FORWARD::preprocess(
	const int P, int D, int M,
	const float* means3D,
	const glm::vec3* scales,
	const float scale_modifier,
	const glm::vec4* rotations,
	const float* opacities,
	const float* shs,
	const float* cov3D_precomp,
	const float* colors_precomp,
	const float* viewmatrix,
	const float* projmatrix,
	const glm::vec3* cam_pos,
	const int W, int H,
	const float focal_x, float focal_y,
	const float tan_fovx, float tan_fovy,
	int* radii,
	bool* clamped,
	float2* means2D,
	float* depths,
	float3* pos_view,
	float* cov3Ds,
	float* rgb,
	float4* conic_opacity,
	uint32_t* tiles_touched,
	const dim3 grid,
	const bool prefiltered,
	const bool cubemap)
{
	preprocessCUDA<NUM_CHANNELS><<<(P + 255) / 256, 256>>> (
		P, D, M,
		means3D,
		scales,
		scale_modifier,
		rotations,
		opacities,
		shs,
		cov3D_precomp,
		colors_precomp,
		viewmatrix, 
		projmatrix,
		cam_pos,
		W, H,
		tan_fovx, tan_fovy,
		focal_x, focal_y,
		clamped,
		radii,
		means2D,
		depths,
		pos_view,
		cov3Ds,
		rgb,
		conic_opacity,
		tiles_touched,
		grid,
		prefiltered,
		cubemap
	);
}

void FORWARD::depthToNormal(
	const dim3 grid,
	const dim3 block,
	const int W, int H,
	const float focal_x,
	const float focal_y,
	const float* viewmatrix,
	const float* depthMap,
	float* normalMap,
	float* normal_from_depth_view) {
	depthmapToNormalCUDA<<<grid, block>>>(
		W, H,
		focal_x,
		focal_y,
		viewmatrix,
		depthMap,
		normalMap,
		normal_from_depth_view
	);
}

void FORWARD::SSAO(
	const dim3 grid, 
	const dim3 block,
	int W, int H,
	const float focal_x,
	const float focal_y,
	const float radius,  //0.8
	const float bias, //-0.01
	const float thick, //-0.05
	const float delta, //0.0625
	const int step, //16
	const int start,
	const float* out_normal,
	const float* out_pos,
	float* occlusion) {
	SSAOCUDA<<<grid, block>>>(
		W, H,
		focal_x,
		focal_y,
		radius,  //0.8
		bias, //-0.01
		thick, //-0.05
		delta, //0.0625
		step, //16
		start,
		out_normal,
		out_pos,
		occlusion
	);
}

void FORWARD::SSR(
	const dim3 grid, 
	const dim3 block,
	int W, int H,
	const float focal_x,
	const float focal_y,
	const float radius,  //0.8
	const float bias, //-0.01
	const float thick, //-0.05
	const float delta, //0.0625
	const int step, //16
	const int start,
	const float* out_normal,
	const float* out_pos,
	const float* out_rgb,
	const float* out_albedo,
    const float* out_roughness,
    const float* out_metallic,
    const float* out_F0,
	float* color,
	float* abd) {
	SSRCUDA<<<grid, block>>>(
		W, H,
		focal_x,
		focal_y,
		radius,  //0.8
		bias, //-0.01
		thick, //-0.05
		delta, //0.0625
		step, //16
		start,
		out_normal,
		out_pos,
		out_rgb,
		out_albedo,
		out_roughness,
		out_metallic,
		out_F0,
		color,
		abd
	);
}

